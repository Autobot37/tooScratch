#include "hip/hip_runtime.h"
//_____________CUDA_MAT_ADD__________________
//https://docs.nvidia.com/cuda/cuda-c-programming-guide/index.html#notices
__global__ void MatAdd(float A[N][N], float B[N][N], float C[N][N]){
    int i = blockIdx.x*blockDim.x + threadIdx.x;
    int j = blockIdx.y*blockDim.y + threadIdx.y;

    if(i<N && j<N){
        C[i][j] = A[i][j] + B[i][j];
    }
}

int main(){

    dim3 threadPerBlock(16,16);
    dim3 numBlocks(N/threadPerBlock.x, N/threadPerBlock.y);
    MatAdd<<<numBlocks,threadPerBlock>>>(A,B,C);


}